#include <hipcub/hipcub.hpp>

extern "C" {
// Sort for integer arrays
void sort_int_wrapper(int *i_data, int *o_data, int N) {
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, i_data,
                                 o_data, N);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, i_data,
                                 o_data, N);
  hipFree(d_temp_storage);
}
}
